#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <getopt.h>

void printDeviceProperties(int deviceId, bool printArch, bool printSMCount) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);

    if (printArch) {
        // Print the CMake compilation instructions
        std::cout << "-gencode;arch=compute_" << prop.major << prop.minor << ",code=sm_" << prop.major << prop.minor;
    }

    if (printSMCount) {
        // Print the number of SMs
        std::cout << prop.multiProcessorCount;
    }
}

int main(int argc, char** argv) {
    int deviceCount;
    // Get the number of available devices
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return -1;
    }

    int opt;
    bool printArch = false;
    bool printSMCount = false;

    // parse command line options
    while ((opt = getopt(argc, argv, "as")) != -1) {
        switch (opt) {
            case 'a':
                printArch = true;
                break;
            case 's':
                printSMCount = true;
                break;
            default:
                std::cerr << "Usage: " << argv[0] << " [-a] [-s]" << std::endl;
                return -1;
        }
    }

    // 打印所有设备的属性
    printDeviceProperties(0, printArch, printSMCount);

    return 0;
}
